#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "asm.h"

#define BLOCK_SIZE 16


__global__ void gpu_matrix_mult(const int *a, const int *b, int *c, int m, int n, int k) {
    unsigned row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < k && row < m) {
        for (int i = 0; i < n; i++) {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

void cpu_matrix_mult(const int *h_a, const int *h_b, int *h_result, int m, int n, int k) {

    omp_set_num_threads(10);
#pragma omp parallel for schedule(dynamic, 10)
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < k; ++j) {
            int tmp = 0;
            for (int h = 0; h < n; ++h) {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}


int main() {
    /* Fixed seed for illustration */
    srand(time(NULL));

    int m = 2000;
    int n = 2000;
    int k = 2000;

    // allocate memory in host RAM, h_cc is used to store CPU result
    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int) * m * n);
    hipHostMalloc((void **) &h_b, sizeof(int) * n * k);
    hipHostMalloc((void **) &h_c, sizeof(int) * m * k);
    hipHostMalloc((void **) &h_cc, sizeof(int) * m * k);

    // random initialize matrix A
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 12;
        }
    }

    // random initialize matrix B
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 12;
        }
    }

    declare_timer
    start_timer

    // Allocate memory space on the device
    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int) * m * n);
    hipMalloc((void **) &d_b, sizeof(int) * n * k);
    hipMalloc((void **) &d_c, sizeof(int) * m * k);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int) * n * k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // Launch kernel
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);
    hipMemcpy(h_c, d_c, sizeof(int) * m * k, hipMemcpyDeviceToHost);

    stop_timer
    printf("GPU: %.2fs\n", (double) elapsed / 1000000.0f);

    start_timer
    cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);
    stop_timer
    printf("CPU: %.2fs\n", (double) elapsed / 1000000.0f);


    // validate results computed by GPU
    int all_ok = 1;
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < k; ++j) {
            if (h_cc[i * k + j] != h_c[i * k + j]) {
                all_ok = 0;
            }
        }
    }

    // roughly compute speedup
    if (!all_ok)printf("incorrect results\n");


    return 0;
}